#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstddef>

#include "cuda_utils.cuh"

// Kernel definition
__global__ void device_add_vectors(const int size, const int *A, const int *B,
                                 int *C) {
  // Calculate id of thread, based on which block it is in and which thread
  // within the block
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < size) {
    C[i] = A[i] + B[i];
  }
}

int* alloc_vector_host(const int size) {
	int* vector;
	check_status(
			hipHostMalloc((void **)&vector, size * sizeof(int)), 
			"alloc vector");
	return vector;
}

void free_vector_host(int* vector) {
	hipHostFree(vector);
}

void add_vectors(const int size, const int *A, const int *B, int *C) {
  // Initialize device (GPU memory) pointers
	GPUArray<int> gpu_A(size), gpu_B(size), gpu_C(size);

	gpu_A.copy_from(A);
	gpu_B.copy_from(B);

  // Number of threads per block
  int blockSize = 1024;
  // Size of grid, number of blocks needed
  int gridSize = (int)std::ceil((float)size / blockSize);

  // Execute calculation
  // NOTICE: You pass the device pointers, not the host pointers
  device_add_vectors<<<gridSize, blockSize>>>(size, gpu_A.get_pointer(), gpu_B.get_pointer(), gpu_C.get_pointer());

	gpu_C.copy_to(C);
}
