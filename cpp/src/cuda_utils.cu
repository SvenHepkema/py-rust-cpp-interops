#include "cuda_utils.cuh"

void check_status(const hipError_t status, const std::string action) {
	if (status != hipSuccess) {
		throw std::runtime_error("CUDA error during \"" + action + "\": " 
				+ std::to_string(status) + " " + hipGetErrorString(status));
	}
}
